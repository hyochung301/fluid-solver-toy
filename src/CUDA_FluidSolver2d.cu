#include "hip/hip_runtime.h"
#include "CUDA_FluidSolver2d.cuh"
#include <flgl/logger.h>
#include <random>
#include <cstring>
#include <hip/hip_runtime.h>
LOG_MODULE(fluid_solver)

#define cu() \
{hipError_t err = hipGetLastError();en++;\
while (err != hipSuccess) {\
    printf("E %d: %s\n", en, hipGetErrorString(err));\
    err = hipGetLastError(); \
}}\

// === private members: ===
// const int n;
// fftwf_plan forward_u, forward_v, inv_u, inv_v;
// float* u, * v, * u0, * v0;
// float visc;
// float* buffer;


__global__ void apply_forces_kernel(float* u, float* v, float* u0, float* v0, float dt, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N*N) {
		u[i] += dt*u0[i]; 
		u0[i] = u[i];
		v[i] += dt*v0[i]; 
		v0[i] = v[i];
	}
}

__global__ void self_advection_kernel(float* u, float* v, float* u0, float* v0, float dt, int N) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int i = index / N;
	int j = index % N;

	if (index < N*N) {
		float x = i - dt * u0[i + N*j] * N;
		float y = j - dt * v0[i + N*j] * N;

		int i0 = floor(x); float s = x - i0; i0 = (N + (i0 % N)) % N; int i1 = (i0 + 1) % N;
		int j0 = floor(y); float t = y - j0; j0 = (N + (j0 % N)) % N; int j1 = (j0 + 1) % N;

		u[i + N*j] = (1 - s) * ((1 - t) * u0[i0 + N*j0] + t * u0[i0 + N*j1]) +
					  s * ((1 - t) * u0[i1 + N*j0] + t * u0[i1 + N*j1]);
		v[i + N*j] = (1 - s) * ((1 - t) * v0[i0 + N*j0] + t * v0[i0 + N*j1]) +
					  s * ((1 - t) * v0[i1 + N*j0] + t * v0[i1 + N*j1]);
	}
}

__global__ void copy_to_fourier_buffers_kernel(float* u, float* v, float* u0, float* v0, int N) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int i = index / N;
	int j = index % N;

	if (index < N*N) {
		u0[2*(i + N*j)] = u[i+N*j]; // Real part of u0
		v0[2*(i + N*j)] = v[i+N*j]; // Real part of v0
		u0[2*(i + 1 + N*j)] = 0.; // Imaginary part of u0
		v0[2*(i + 1 + N*j)] = 0.; // Imaginary part of v0
	}
}

__global__ void applyLowPassFilter(float* u0, float* v0, float dt, float visc, int N) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int i = index / N;
	int j = index % N;

	if (index < N * N) {
		float x = (i <= N/2) ? i : (float)i - (float)N;
		float y = (j <= N/2) ? j : (float)j - (float)N;
		float r = x*x + y*y;
		if (r != 0.0) {
			float *uf = &(u0[2*(i + N*j)]);
			float *vf = &(v0[2*(i + N*j)]);

			float f = exp(-r * dt * visc);

			float ur = f * ( (1 - x*x/r)*uf[0] - x*y/r * vf[0] );
			float ui = f * ( (1 - x*x/r)*uf[1] - x*y/r * vf[1] );
			float vr = f * ( -y*x/r * uf[0] + (1 - y*y/r)*vf[0] );
			float vi = f * ( -y*x/r * uf[1] + (1 - y*y/r)*vf[1] );

			uf[0] = ur;
			uf[1] = ui;
			vf[0] = vr;
			vf[1] = vi;
		}
	}
}

__global__ void normalizeAndClear(float* u, float* v, float* u0, float* v0, float f, int N) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int i = index / N;
	int j = index % N;

	if (index < N * N) {
		u[i+N*j] = f * u0[2*(i + N*j)]; // Real part of u0
		v[i+N*j] = f * v0[2*(i + N*j)]; // Real part of v0

		u0[2*(i + N*j)] = 0.; // Clear u0
		v0[2*(i + N*j)] = 0.; // Clear v0
	}
}

StamFFT_FluidSolver::StamFFT_FluidSolver(int const& N) : n(N),
                                                         viscosity(visc),
                                                         force_mul(1.),
                                                         timer(MICROSECONDS) {
    LOG_DBG("constructing fluid solver...");
	visc = 0.001;
    alloc_buffers();
}
StamFFT_FluidSolver::~StamFFT_FluidSolver() {
    free_buffers();
}

void StamFFT_FluidSolver::use_ffts(FFT_Solver2d* fu, FFT_Solver2d* fv) {
    fftu = fu; fftv = fv;
}

float* StamFFT_FluidSolver::buff() const {return buffer;}
float* StamFFT_FluidSolver::x_buffer() const {return u;}
float* StamFFT_FluidSolver::y_buffer() const {return v;}
float* StamFFT_FluidSolver::fx_buffer() const {return u0;}
float* StamFFT_FluidSolver::fy_buffer() const {return v0;}
int const& StamFFT_FluidSolver::dim() const {return n;}

void StamFFT_FluidSolver::add_force(int x, int y, int fx, int fy) {
    if (!((x+y*n) < n*n) || (x < 0) || (y < 0) || (x >= n) || (y >= n)) {return;}//LOG_ERR("add force at %d,%d = idx %d OOB", x, y, x+y*n); return;}
	u0[x+y*n] += fx * force_mul;
	v0[x+y*n] += fy * force_mul;
}

void StamFFT_FluidSolver::set_force(int x, int y, int fx, int fy) {
    if (!((x+y*n) < n*n) || (x < 0) || (y < 0) || (x >= n) || (y >= n)) {return;}//LOG_ERR("add force at %d,%d = idx %d OOB", x, y, x+y*n); return;}
	u0[x+y*n] = fx * force_mul;
	v0[x+y*n] = fy * force_mul;
}

void StamFFT_FluidSolver::get_force(int x, int y, int& fx, int& fy) const {
    if (!((x+y*n) < n*n) || (x < 0) || (y < 0) || (x >= n) || (y >= n)) {return;}//LOG_ERR("add force at %d,%d = idx %d OOB", x, y, x+y*n); return;}
	fx = u0[x+y*n] / force_mul;
	fy = v0[x+y*n] / force_mul;
}

void StamFFT_FluidSolver::set_force_multiplier(float mul) {
    force_mul = mul;
}

float StamFFT_FluidSolver::force_multiplier() {
    return force_mul;
}

void StamFFT_FluidSolver::alloc_buffers() {
    int k;
    u = new float[n*n];
    v = new float[n*n];
    for (k = 0; k < n*n; k++) {u[k]=0.;v[k]=0.;}
    u0 = new float[n*n*2];
    v0 = new float[n*n*2];
    for (k = 0; k < n*n*2; k++) {u0[k]=0.;v0[k]=0.;}
    buffer = new float[n*n*2];
    for (k = 0; k < n*n*2; k++) {buffer[k]=0.;}
}

void StamFFT_FluidSolver::free_buffers() {
    delete [] u;
    delete [] v;
    delete [] u0;
    delete [] v0;
    delete [] buffer;
}

void StamFFT_FluidSolver::random_fill(float mag) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-mag, mag);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            u0[2*(i+j*n)] = dis(gen); v0[2*(i+j*n)] = dis(gen);
        }
    }
}

void StamFFT_FluidSolver::zero_field() {
    memset(u,0,sizeof(float)*n*n);
    memset(v,0,sizeof(float)*n*n);
}

void StamFFT_FluidSolver::slow_fill_pixbuff() {
	float buffmax = -9999999.f;
	float buffmin = 9999999.f;
	for (int i = 0; i < n; i++) {
		int x = i - (n/2);
		for (int j = 0; j < n; j++) {
			int y = j - (n/2);
			int r = 2*(i+j*n);
			int g = 2*(i+j*n) + 1;
			buffer[r] = u[i+j*n];
			buffer[g] = v[i+j*n];
			if (buffer[r]>buffmax) buffmax = buffer[r];
			if (buffer[g]>buffmax) buffmax = buffer[g];
			if (buffer[r]<buffmin) buffmin = buffer[r];
			if (buffer[g]<buffmin) buffmin = buffer[g];
		}
	}
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			int r = 2*(i+j*n);
			int g = 2*(i+j*n) + 1;
			float imag = (1.f / ((buffmax-buffmin)==0.?1.f:(buffmax-buffmin)));
			buffer[r] -= buffmin; buffer[r] *= imag;
			buffer[g] -= buffmin; buffer[g] *= imag;
		}
	}
}

void StamFFT_FluidSolver::step(float const& dt) {
	this->stam_stable_solve(n, u, v, u0, v0, visc, dt);
}

float StamFFT_FluidSolver::get_prev_solver_t() {
    return t_us_solver;
}

float StamFFT_FluidSolver::get_prev_fft_t() {
    return t_us_ffts;
}

#define TIMER_ACCUMULATION_START(grbg)  st = timer.read();
#define TIMER_ACCUMULATION_END(grbg)    en = timer.read(); t_us_ffts += en-st;

#define __floor(x) ((x)>=0.0?((int)(x)):(-((int)(1-(x)))))

/*
	This is Stam's solver itself
	It is almost the same as in his paper
	but adapted to use fftw3 and commented

	Inputs:
		n: 			size of n by n field
		(u, v): 	velocity of the prev time step
		(u0, v0): 	a force field defined on a grid
		visc: 		viscosity of fluid
		dt: 		time step
	Outputs:
		(u, v): 	next step of fluid solver
		(u0,v0):	cleared for new forces
*/
#define BUFF_R(buf, i, j) buf[2*(i  +(j*N))]
#define BUFF_I(buf, i, j) buf[2*(i+1+(j*N))]

void StamFFT_FluidSolver::stam_stable_solve(int const& N,
                       float* const u, float* const v,
                       float* const u0, float* const v0,
                       float const& visc, float const& dt)
{
    float x, y, f, r, U[2], V[2], s, t;
    int i, j, i0, j0, i1, j1;
	float *dev_u, *dev_v, *dev_u0, *dev_v0;

	// grid size
	dim3 blockSize(256);
	dim3 gridSize((N * N + blockSize.x - 1) / blockSize.x);

    // debug code
    timer.reset_start(); float st; float en; st=en=t_us_ffts=t_us_solver=0.;


	// Assuming u, v, u0, and v0 are already allocated and initialized on the host
	hipMalloc((void**)&dev_u, N * N * sizeof(float));
	hipMalloc((void**)&dev_v, N * N * sizeof(float));
	hipMalloc((void**)&dev_u0, N * N * 2 * sizeof(float)); // Times 2 for complex numbers (real and imaginary parts)
	hipMalloc((void**)&dev_v0, N * N * 2 * sizeof(float)); // Same as above

	// Copy data from host to device
	hipMemcpy(dev_u, u, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_v, v, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_u0, u0, N * N * 2 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_v0, v0, N * N * 2 * sizeof(float), hipMemcpyHostToDevice);


    // apply forces
	apply_forces_kernel<<<gridSize, blockSize>>>(dev_u, dev_v, dev_u0, dev_v0, dt, N);
	hipDeviceSynchronize()

    // self-advection: semi-Lagrangian scheme
    // here, (u0,v0) are used to interpolate
    // and (u,v) stores the interpolation & result
	self_advection_kernel<<<gridSize, blockSize>>>(dev_u, dev_v, dev_u0, dev_v0, dt, N);
	hipDeviceSynchronize();

    // copy velos into real-part of fourier buffers (i think)
	copy_to_fourier_buffers_kernel<<<gridSize, blockSize>>>(dev_u, dev_v, dev_u0, dev_v0, N);
	hipDeviceSynchronize();

    // transform to fourier domain
    TIMER_ACCUMULATION_START(); // debug code
    fftu->forward(); fftv->forward();           // SPATIAL -> FOURIER
    TIMER_ACCUMULATION_END();   // debug code

    // apply low pass filter to simulate viscosity
    // and force field to be mass converving
    // by projecting vectors onto line perpendicular to wave #
    // which is line tan to circles centered at origin
	applyLowPassFilter<<<gridSize, blockSize>>>(dev_u0, dev_v0, dt, visc, N);
	hipDeviceSynchronize();

    // inverse ffts back to spatial domain
    TIMER_ACCUMULATION_START(); // debug code
    fftu->inverse(); fftv->inverse();           // FOURIER -> SPATIAL
    TIMER_ACCUMULATION_END();   // debug code

    // normalize (r2c then c2r tform multiplies all by n*n)
	float f = 1.0/(N*N);
	normalizeAndClear<<<gridSize, blockSize>>>(dev_u, dev_v, dev_u0, dev_v0, 1.0f / (N * N), N);
	hipDeviceSynchronize();

    // clear force field
    memset(u0,0,sizeof(float)*N*N*2);
    memset(v0,0,sizeof(float)*N*N*2);

	hipMemcpy(u, dev_u, N * N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(v, dev_v, N * N * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dev_u);
	hipFree(dev_v);
	hipFree(dev_u0);
	hipFree(dev_v0);

    t_us_solver = timer.stop();
}

